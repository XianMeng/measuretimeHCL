/* Aug. 2019
*
*  gpu.cu: GPU part
*/

#include <stdio.h>
#include <sys/time.h>
#include <assert.h>
#include "cpu.h"

// Error handling macro
#define CUDA_CHECK(call) \
    if((call) != hipSuccess) { \
        hipError_t err = hipGetLastError(); \
        fprintf(stderr,"CUDA error calling \""#call"\", code is %d\n",err); \
        my_abort(err); }
//#define DEBUG
#define _MIC_

// 1.Initialize an array with all 1 on cpu
// 2. Copy the data from cpu to gpu
// 3. Copy the data from cpu to gpu
//void * initData(int * data_in_cpu, int * data_from_gpu, int dataSize) {
//int * gpu0_to_cpu0(int * data_in_cpu, int * data_from_gpu, int dataSize) {
void gpu0_to_cpu0(int * data_in_cpu, int * data_from_gpu, int dataSize) {
    //Recording time
    struct timeval start, end;
    double GPU_to_CPU_time;
    double tstart , tend;

    //Data in GPU
    int * gpu_data_from_cpu = NULL; //data from cpu to gpu
    hipMalloc((void**)&gpu_data_from_cpu, dataSize * sizeof(int));  

    // Allocate pinned host memory
    hipHostAlloc(&data_in_cpu, sizeof(int) * dataSize, hipHostMallocDefault);
    hipHostAlloc(&data_from_gpu, sizeof(int) * dataSize, hipHostMallocDefault);
    #ifdef _MIC_
    hipHostAlloc(&phi_data, sizeof(int) * dataSize, hipHostMallocDefault);
    #endif

    // Initialize host data
    for(int i = 0; i < dataSize; i++) {
        data_in_cpu[i] = 1;
        #ifdef _MIC_
        phi_data[i]=1;
        #endif
    }
   
    //Copy data from cpu to gpu
    hipMemcpy(gpu_data_from_cpu, data_in_cpu, sizeof(int) * dataSize, hipMemcpyHostToDevice);
    
    // Allocate PHI memory
    #ifdef _MIC_
    #pragma offload_transfer target(mic) \
	in( phi_data : length(dataSize) ALLOC 
    #endif

    //Transfer data from GPU to CPU memory, and record time
    gettimeofday(&start, NULL);
    hipMemcpy(data_from_gpu, gpu_data_from_cpu, sizeof(int) * dataSize, hipMemcpyDeviceToHost);
    gettimeofday(&end, NULL);

    //Transfer data from host to PHY
    #ifdef _MIC_
    #pragma offload_transfer target(mic) in( phi_data : length(dataSize)  REUSE )
    #endif

    tstart = start.tv_sec + start.tv_usec/1000000.;
    tend = end.tv_sec + end.tv_usec/1000000.;
    GPU_to_CPU_time = (tend - tstart);
    printf("############################ GPU to CPU transfer time  ##############################\n");
    printf(" %d int data,      time is:%f seconds\n",dataSize, GPU_to_CPU_time);
    printf("#####################################################################################\n");

    #ifdef DEBUG
    printf( "data_in_cpu[1] = %d\n", data_in_cpu[1]);
    printf( "data_from_gpu[1] = %d\n",data_from_gpu[1]);
    #endif
 
    //Free GPU memory
    CUDA_CHECK(hipFree(gpu_data_from_cpu)); 

    //Free host memory
    CUDA_CHECK(hipHostFree(data_in_cpu));

    //Deallocate PHI memory
    #ifdef
    #pragma offload_transfer target(mic) \
        in( phi_data : length(dataSize) FREE  ), \
        in ( cpu_data : length(144728064) FREE )
    #endif
}

void cpu1_to_gpu1(int * cpu1_data, int dataSize){
   //Recoding time
   struct timeval start, end;
   double CPU_to_GPU_time;
   double tstart , tend;

   //data in cpu;
   int * data_in_cpu;

   //Data in GPU
   int * gpu_data_from_cpu = NULL; //data from cpu to gpu
   CUDA_CHECK(hipMalloc((void**)&gpu_data_from_cpu, dataSize * sizeof(int)));

   // Allocate pinned host memory
   hipHostAlloc(&data_in_cpu, sizeof(int) * dataSize, hipHostMallocDefault);

   int i;
   for (i=1;i<dataSize;i++)
     {
     data_in_cpu[i]=cpu1_data[i];
     }
 
   //Copy data from cpu to gpu
   gettimeofday(&start, NULL);
   hipMemcpy(gpu_data_from_cpu, data_in_cpu, sizeof(int) * dataSize, hipMemcpyHostToDevice); 
   gettimeofday(&end, NULL);

   tstart = start.tv_sec + start.tv_usec/1000000.;
   tend = end.tv_sec + end.tv_usec/1000000.;
   CPU_to_GPU_time = (tend - tstart);

   printf("############################ CPU to GPU transfer time  ##############################\n");
   printf(" %d int data,      time is:%f seconds\n",dataSize, CPU_to_GPU_time);
   printf("#####################################################################################\n");

   //Free GPU memory
    CUDA_CHECK(hipFree(gpu_data_from_cpu));

   //Free host memory
   CUDA_CHECK(hipHostFree(data_in_cpu));
   }

   //hipDeviceReset();

